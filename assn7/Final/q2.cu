
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>
#include<math.h>
#define NUM 10000000
#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);

#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))

__device__ unsigned int exor(unsigned long a,unsigned long b)
{	
	unsigned int res;

	   for (int i = 63; i >= 0; i--)                     
    {
       // Find current bits in x and y
       bool b1 = a & (1 << i);
       bool b2 = b & (1 << i);
        
        // If both are 1 then 0 else xor is same as OR
        bool xoredBit = (b1 & b2) ? 0 : (b1 | b2);          
 
        // Update result
        res <<= 1;
        res |= xoredBit;
    }
	//res=exor(a,b);
	return res;
}
__global__ void calculate(unsigned long *mem,unsigned long num,int l,unsigned long space)
{
      int i = blockDim.x * blockIdx.x + threadIdx.x;
      if(i >= num/2*(l+1))
           return;
      unsigned long tmp=i*2*space;
      if(tmp+space<num)
       	 mem[tmp]=exor(mem[tmp],mem[tmp+space]);
}

int main(int argc, char **argv){
	struct timeval start, end, t_start, t_end;
    int i,blocks=0;
	unsigned long *p1,*g1; 
    unsigned long seed,num;

    if(argc == 3){
    num = atoi(argv[1]);   /*Update after checking*/
    if(num <= 0)
       num = NUM;
    seed=atoi(argv[2]);
    }
    p1 = (unsigned long *)malloc((num+1) *sizeof(unsigned long));
    srand(seed); 
    for(i=0; i<num; ++i){
 	p1[i]=random();
    }
    p1[i]=0;

    gettimeofday(&t_start, NULL);
    hipMalloc(&g1, (num+1) * sizeof(unsigned long));
    CUDA_ERROR_EXIT("hipMalloc");

    hipMemcpy(g1, p1, (num+1) * sizeof(unsigned long) , hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("hipMemcpy");
    
    gettimeofday(&start, NULL); 
	
   blocks=num/1024;
   if(num%1024)
	++blocks;

    for(i=0;i<log(num)/log(2);i++){
        calculate<<<blocks,1024>>>(g1,num,i,(unsigned long)pow(2,i));
   }



    CUDA_ERROR_EXIT("kernel invocation");
    gettimeofday(&end, NULL);
    
    /* Copy back result*/

    hipMemcpy(p1, g1, (num+1) * sizeof(unsigned long), hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("memcpy");
    gettimeofday(&t_end, NULL);
    
   printf("The Final XOR Value is %lu\n",p1[0]);
   printf("Total time = %ld microsecs Processsing =%ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));
    hipFree(g1);
    free(p1);
	return 0;
}


