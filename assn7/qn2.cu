
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>
#include<math.h>
#define NUM 10000000
#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);

#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))

__device__ unsigned int exor(unsigned long a,unsigned long b)
{	
	unsigned int res;

	   for (int i = 63; i >= 0; i--)                     
    {
       // Find current bits in x and y
       bool b1 = a & (1 << i);
       bool b2 = b & (1 << i);
        
        // If both are 1 then 0 else xor is same as OR
        bool xoredBit = (b1 & b2) ? 0 : (b1 | b2);          
 
        // Update result
        res <<= 1;
        res |= xoredBit;
    }
	//res=exor(a,b);
	return res;
}
__global__ void calculate(unsigned long *mem,int num,int iter)
{
      int i = blockDim.x * blockIdx.x + threadIdx.x;
      if(i >= num)
           return;
    // unsigned long *t1,*t2;
if(i<num/2){
        int tmp=i*2;
	if(tmp+iter<num)
        	mem[tmp]=exor(mem[tmp],mem[tmp+iter]);
//	else
	      // mem[tmp]=exor(mem[tmp],0);
    // mem[num]=res;
}
}

int main(int argc, char **argv){
	struct timeval start, end, t_start, t_end;
    int i,blocks=0;
	unsigned long *p1,*g1; 
    unsigned long seed,num;

    if(argc == 3){
    num = atoi(argv[1]);   /*Update after checking*/
    if(num <= 0)
       num = NUM;
    seed=atoi(argv[2]);
    }
//    printf("%d",time(0));
    p1 = (unsigned long *)malloc((num+1) *sizeof(unsigned long));
    srand(seed); 
    for(i=0; i<num; ++i){
 	p1[i]=random();
//	printf("%d  %lu\n",i,p1[i]);
    }
    p1[i]=0;

    gettimeofday(&t_start, NULL);

//for(i=0;i<num;i++)
//	printf("%d-%lu\n",i,p1[i]);
    hipMalloc(&g1, (num+1) * sizeof(unsigned long));
    CUDA_ERROR_EXIT("hipMalloc");

    hipMemcpy(g1, p1, (num+1) * sizeof(unsigned long) , hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("hipMemcpy");
    
    gettimeofday(&start, NULL);
    blocks = num /1024;
    
    if(num % 1024)
           ++blocks;
	   


for(i=0;i<log(num)/log(2);i++){
    calculate<<<blocks,1024>>>(g1,num,(int)pow(2,i));
}



    CUDA_ERROR_EXIT("kernel invocation");
    gettimeofday(&end, NULL);
    
    /* Copy back result*/

    hipMemcpy(p1, g1, (num+1) * sizeof(unsigned long), hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("memcpy");
    gettimeofday(&t_end, NULL);
    
//   for(i=0;i<num;i++)
//	printf("%d-%lu\n",i,p1[i]);   

   printf("%lu",p1[0]);
 printf("Total time = %ld microsecs Processsing =%ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));
    hipFree(g1);
   
    /*Print the last element for sanity check*/ 
    printf("The XOR final value is %lu",p1[num]);
    
    free(p1);
 //   free(pfinal);

	return 0;
}


